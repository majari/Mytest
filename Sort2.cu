#include<thrust/sort.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<thrust/device_ptr.h>

int main()
{
   const int N = 6;
   int keys_h[N] = {  1,   2,   1,   2,   4,   1};
   int values_h[N] = {10,20,300,400,600,200};
   int *keys_d,*values_d; 
	
	size_t size = N * sizeof(int);
 
	hipMalloc((void **) &keys_d, size);
	hipMalloc((void **) &values_d, size);
	 
	hipMemcpy(keys_d, keys_h, size, hipMemcpyHostToDevice);
	hipMemcpy(values_d, values_h, size, hipMemcpyHostToDevice);

  	thrust::sort_by_key(keys_h, keys_h + N, values_h);
	
	hipMemcpy(keys_d, keys_h, size, hipMemcpyDeviceToHost);
	hipMemcpy(values_d, values_h, size, hipMemcpyDeviceToHost);

 	for(int i=0; i<N; i++)printf("%d %d \n",keys_h[i],values_h[i]);
	
	free(keys_h); hipFree(keys_d);
	free(values_h); hipFree(values_d);
 
	 return 0;
}
