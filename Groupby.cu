#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<assert.h>
#define N 10 // side of matrix containing data



__global__ void sum_value(float * da,  float *result)
{
	 int tid = blockDim.x * blockIdx.x + threadIdx.x;
	 
	 if(tid < N){
		 float sum=0.0;
	       	 for(int i=0;i < N ; i++)
		 {
		        //Each thread does sum of its row, thread 0 => 0 to N-1 , thread 1 => n to 2N-1 elements;	  	
			sum= sum+da[N*tid+i];
			//printf("thread= %d, da[%d]= %f\n",tid,N*tid+i,da[N*tid+i]);
		 } 
		 //printf("thread= %d, sum= %f\n",tid,sum);
		 syncthreads();
		 result[tid]= sum;
	 }
}
__global__ void min_key(float *da, float * result)
{
	int tid= blockDim.x*blockIdx.x+threadIdx.x;
	if(tid < N)
	{	
		float minkey=da[N*tid]; // initially first elememt as min
		for(int i=0; i< N; i++){
			if(minkey > da[N*tid+i])
				minkey = da[N*tid+i];	
		}
		syncthreads();
		result[tid] = minkey;
	}
}

__global__ void max_key(float *da, float * result)
{
	int tid= blockDim.x*blockIdx.x+threadIdx.x;
	if(tid < N)
	{	
		float maxkey=da[N*tid]; // initially first element as max
		for(int i=0; i< N; i++){
			if(maxkey < da[N*tid+i])
				maxkey= da[N*tid+i];	
		}
		syncthreads();
		result[tid] = maxkey;
	}
}

	
void init(float * array, int size);
void check_result(float * result);
void check_minresult(float * result);
void check_maxresult(float * result);
	 

int main(int argc, char * argv[])

{
	float * A, *dA,  *dresult, *dminresult, *dmaxresult,*hresult, *hminresult, *hmaxresult;

	A = (float *)malloc(sizeof(float)*N*N);
	hresult = (float *)malloc(sizeof(float)*N);
	hminresult = (float *)malloc(sizeof(float)*N);
	hmaxresult = (float *)malloc(sizeof(float)*N);
	
	init(A,N*N);

	hipMalloc((void **)&dA,sizeof(float)*N*N);	 
	hipMalloc((void **)&dresult, sizeof(float)*N);
	hipMalloc ((void **)&dminresult, sizeof(float)*N);
	hipMalloc((void **)&dmaxresult, sizeof(float)*N);

	 //copy memory from unpadded array A of 10 cross 10 dimensions
	 hipMemcpy(dA,A,sizeof(float)*N*N,hipMemcpyHostToDevice);
         // launching kernel, total number of threads = total number of keys / rows => number of blocks* number of threads	
	 sum_value<<< 1,N >>>(dA,dresult);
	 min_key <<< 1, N >>>(dA,dminresult);
	 max_key<<< 1, N>>>(dA, dmaxresult);
	 
         hipMemcpy(hresult,dresult,sizeof(float)*N,hipMemcpyDeviceToHost);
         hipMemcpy(hminresult,dminresult,sizeof(float)*N,hipMemcpyDeviceToHost);
	 hipMemcpy(hmaxresult,dmaxresult,sizeof(float)*N,hipMemcpyDeviceToHost);
  
	 check_result(hresult);
	 check_minresult(hminresult);
	 check_maxresult(hmaxresult);
 	 free(A);
	 hipFree(dA);
}
	 
void check_result(float * result){
	for(int i=0;i<N; i++){
		printf(" Key %d -> Sum= %f\n", i, result[i]);
	}	
}

void check_minresult(float * result){
	for(int i=0;i<N; i++){
		printf(" Key %d -> MinKey= %f\n", i, result[i]);
	}	
}

void check_maxresult(float * result){
	for(int i=0;i<N; i++){
		printf(" Key %d -> MaxKey= %f\n", i, result[i]);
	}	
}

void init(float * array, int size)
{
	for (int i = 0; i < N*N; i++)

	 {
	 	array[i] = (float)i;
		//printf("Init=> %d = %f\n",i,array[i]);
	 }
}

