#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<thrust/sort.h>
#include<thrust/device_ptr.h>

int main(void)
{
	int *a_h,*a_d;
	const int N=10;
	size_t size = N*sizeof(int);
	a_h = (int*)malloc(size);
	hipMalloc((void**) &a_d ,size);
	std::cout<<"enter 10 numbers";

	for(int i =0; i<N; i++)
	{
		std::cin>>a_h[i];
	}
	for(int i =0; i<N; i++)
	{ 
		
		printf("%d %d\n", i, a_h[i]);
	}
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	thrust::device_ptr<int> t_a(a_d);
	thrust::sort(t_a, t_a + N);

	hipMemcpy(a_h, a_d, sizeof(int)*N,hipMemcpyDeviceToHost);
	
	printf("Sorted list is:\n");
	
	for(int i=0; i<N; i++)
	{ 
		
		printf("%d %d\n", i, a_h[i]);
	}

	free(a_h); hipFree(a_d);
	
	return 0;
}
